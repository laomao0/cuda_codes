#include "hip/hip_runtime.h"

/* asum: sum of all entries of a vector */

#include "reduction_aux.h"

__device__ void warpReduce(volatile FLOAT *sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

/* sum all entries in x and asign to y
 * block dim must be 256 */
__global__ void asum_stg_1(const FLOAT *x, FLOAT *y, int N)
{
    __shared__ FLOAT sdata[256];
    int idx = get_tid();
    int tid = threadIdx.x;
    int bid = get_bid();

    /* load data to shared mem */
    if (idx < N) {
        sdata[tid] = x[idx];
    }
    else {
        sdata[tid] = 0;
    }

    __syncthreads();

    /* reduction using shared mem */
    if (tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();

    if (tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    if (tid < 32) warpReduce(sdata, tid);

    if (tid == 0) y[bid] = sdata[0];
}

__global__ void asum_stg_3(FLOAT *x, int N)
{
    __shared__ FLOAT sdata[128];
    int tid = threadIdx.x;
    int i;

    sdata[tid] = 0;

    /* load data to shared mem */
    for (i = 0; i < N; i += 128) {
        if (tid + i < N) sdata[tid] += x[i + tid];
    }

    __syncthreads();

    /* reduction using shared mem */
    if (tid < 64) sdata[tid] = sdata[tid] + sdata[tid + 64];
    __syncthreads();

    if (tid < 32) warpReduce(sdata, tid);

    if (tid == 0) x[0] = sdata[0];
}

/* dy and dz serve as cache: result stores in dz[0] */
void asum(FLOAT *dx, FLOAT *dy, FLOAT *dz, int N)
{
    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid = dim3(s, s);
    int gs = 0;

    /* stage 1 */
    asum_stg_1<<<grid, bs>>>(dx, dy, N);

    /* stage 2 */
    {
        /* 1D grid */
        int N2 = (N + bs - 1) / bs;

        int s2 = ceil(sqrt((N2 + bs - 1.) / bs));
        dim3 grid2 = dim3(s2, s2);

        asum_stg_1<<<grid2, bs>>>(dy, dz, N2);

        /* record gs */
        gs = (N2 + bs - 1.) / bs;
    }

    /* stage 3 */
    asum_stg_3<<<1, 128>>>(dz, gs);
}

int main(int argc, char **argv)
{
    int N = 10000070;
    int nbytes = N * sizeof(FLOAT);

    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL, *dz;
    int i, itr = 20;
    FLOAT asd = 0, ash;
    double td, th;

    if (argc == 2) {
        int an;

        an = atoi(argv[1]);
        if (an > 0) N = an;
    }

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, sizeof(FLOAT) * ((N + 255) / 256));
    hipMalloc((void **)&dz, sizeof(FLOAT) * ((N + 255) / 256));

    if (dx == NULL || dy == NULL || dz == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %e MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);

    if (hx == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %e MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* let dust fall */
    hipDeviceSynchronize();
    td = get_time();

    /* call GPU */
    for (i = 0; i < itr; i++) asum(dx, dy, dz, N);

    /* let GPU finish */
    hipDeviceSynchronize();
    td = get_time() - td;

    th = get_time();
    for (i = 0; i < itr; i++) ash = asum_host(hx, N);
    th = get_time() - th;

    /* copy data from GPU */
    hipMemcpy(&asd, dz, sizeof(FLOAT), hipMemcpyDeviceToHost);

    printf("asum, answer: %d, calculated by GPU:%f, calculated by CPU:%f\n", N, asd, ash);
    printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);

    hipFree(dx);
    hipFree(dy);
    hipFree(dz);
    free(hx);

    return 0;
}
