#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void AddInstsCUDA(int *a, int *b)
{
    a[0] += b[0];
}

int main()
{
    int a =5, b = 9;
    int *d_a, *d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddInstsCUDA<<<1, 1>>>(d_a, d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    cout << "The answer is " << a << endl;

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}