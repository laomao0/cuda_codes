#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

typedef double FLOAT;

__global__ void sum(FLOAT *x)
{
    int tid = threadIdx.x;

    x[tid] += 1;
}

int main()
{
    int N = 32;
    int nbytes = N * sizeof(FLOAT);

    FLOAT *dx = NULL, *hx = NULL;
    int i;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);

    if (dx == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    /* alllocate CPU host mem: memory copy is faster than malloc */
    hipHostMalloc((void **)&hx, nbytes);

    if (hx == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }

    /* init */
    printf("hx original: \n");
    for (i = 0; i < N; i++) {
        hx[i] = i;

        printf("%g\n", hx[i]);
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    sum<<<1, N>>>(dx);

    /* let GPU finish */
    hipDeviceSynchronize();

    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);

    printf("\nhx from GPU: \n");
    for (i = 0; i < N; i++) {
        printf("%g\n", hx[i]);
    }

    hipFree(dx);
    hipHostFree(hx);

    return 0;
}
