#include "hip/hip_runtime.h"

/* dot product of two vectors: d = <x, y> */

#include "reduction_aux.h"
#include <assert.h>

/* host, add */
FLOAT dot_host(FLOAT *x, FLOAT *y, int N)
{
    int i;
    FLOAT t = 0;

    assert(x != NULL);
    assert(y != NULL);

    for (i = 0; i < N; i++) t += x[i] * y[i];

    return t;
}

__device__ void warpReduce(volatile FLOAT *sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

/* partial dot product */
__global__ void dot_stg_1(const FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    __shared__ FLOAT sdata[256];
    int idx = get_tid();
    int tid = threadIdx.x;
    int bid = get_bid();

    /* load data to shared mem */
    if (idx < N) {
        sdata[tid] = x[idx] * y[idx];
    }
    else {
        sdata[tid] = 0;
    }

    __syncthreads();

    /* reduction using shared mem */
    if (tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();

    if (tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    if (tid < 32) warpReduce(sdata, tid);

    if (tid == 0) z[bid] = sdata[0];
}

/* sum all entries in x and asign to y
 * block dim must be 256 */
__global__ void dot_stg_2(const FLOAT *x, FLOAT *y, int N)
{
    __shared__ FLOAT sdata[256];
    int idx = get_tid();
    int tid = threadIdx.x;
    int bid = get_bid();

    /* load data to shared mem */
    if (idx < N) {
        sdata[tid] = x[idx];
    }
    else {
        sdata[tid] = 0;
    }

    __syncthreads();

    /* reduction using shared mem */
    if (tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();

    if (tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    if (tid < 32) warpReduce(sdata, tid);

    if (tid == 0) y[bid] = sdata[0];
}

__global__ void dot_stg_3(FLOAT *x, int N)
{
    __shared__ FLOAT sdata[128];
    int tid = threadIdx.x;
    int i;

    sdata[tid] = 0;

    /* load data to shared mem */
    for (i = 0; i < N; i += 128) {
        if (tid + i < N) sdata[tid] += x[i + tid];
    }

    __syncthreads();

    /* reduction using shared mem */
    if (tid < 64) sdata[tid] = sdata[tid] + sdata[tid + 64];
    __syncthreads();

    if (tid < 32) warpReduce(sdata, tid);

    if (tid == 0) x[0] = sdata[0];
}

/* dz and d serve as cache: result stores in d[0] */
void dot_device(FLOAT *dx, FLOAT *dy, FLOAT *dz, FLOAT *d, int N)
{
    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid = dim3(s, s);
    int gs = 0;

    /* stage 1 */
    dot_stg_1<<<grid, bs>>>(dx, dy, dz, N);

    /* stage 2 */
    {
        /* 1D grid */
        int N2 = (N + bs - 1) / bs;

        int s2 = ceil(sqrt((N2 + bs - 1.) / bs));
        dim3 grid2 = dim3(s2, s2);

        dot_stg_2<<<grid2, bs>>>(dz, d, N2);

        /* record gs */
        gs = (N2 + bs - 1.) / bs;
    }

    /* stage 3 */
    dot_stg_3<<<1, 128>>>(d, gs);
}

int main(int argc, char **argv)
{
    int N = 10000070;
    int nbytes = N * sizeof(FLOAT);

    FLOAT *hx = NULL, *hy = NULL;
    FLOAT *dx = NULL, *dy = NULL, *dz = NULL, *d = NULL;
    int i, itr = 20;
    FLOAT asd = 0, ash;
    double td, th;

    if (argc == 2) {
        int an;

        an = atoi(argv[1]);
        if (an > 0) N = an;
    }

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);

    hipMalloc((void **)&dz, sizeof(FLOAT) * ((N + 255) / 256));
    hipMalloc((void **)&d, sizeof(FLOAT) * ((N + 255) / 256));

    if (dx == NULL || dy == NULL || dz == NULL || d == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %e MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);

    if (hx == NULL || hy == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %e MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 2;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);

    /* let dust fall */
    hipDeviceSynchronize();
    td = get_time();

    /* call GPU */
    for (i = 0; i < itr; i++) dot_device(dx, dy, dz, d, N);

    /* let GPU finish */
    hipDeviceSynchronize();
    td = get_time() - td;

    th = get_time();
    for (i = 0; i < itr; i++) ash = dot_host(hx, hy, N);
    th = get_time() - th;

    /* copy data from GPU */
    hipMemcpy(&asd, d, sizeof(FLOAT), hipMemcpyDeviceToHost);

    printf("dot, answer: %d, calculated by GPU:%f, calculated by CPU:%f\n", 2 * N, asd, ash);
    printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);

    hipFree(dx);
    hipFree(dy);
    hipFree(dz);
    hipFree(d);

    free(hx);
    free(hy);

    return 0;
}
