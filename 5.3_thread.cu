#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

/*定义 const 指针(由于指针本身的值不能改变所以必须得初始化）*/
__global__ void what_is_my_id(unsigned int * const block,
		unsigned int * const thread,
		unsigned int * const warp,
		unsigned int * const calc_thread)
{
	/* Thread is is block index * block size + thread offset into the block */
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	block[thread_idx] = blockIdx.x;
	thread[thread_idx]= threadIdx.x;

	/* Calculate warp using built in variable warpSize */
	warp[thread_idx] = threadIdx.x / warpSize;

	calc_thread[thread_idx] = thread_idx;

}

/* Declare statically four arrays of ARRAY_SIZE each */

unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];

int main(void)
{
	/* Total thread count = 2 * 64 = 128 */
	const unsigned int num_blocks = 2;
	const unsigned int num_threads = 64;

	// Declare pointers fro GPU based params
	unsigned int * gpu_block;
	unsigned int * gpu_thread;
	unsigned int * gpu_warp;
	unsigned int * gpu_cal_thread;

	// Declare loop counter for use later
	unsigned int i;

	// Allocate four arrays on the GPU
	hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_warp, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_cal_thread, ARRAY_SIZE_IN_BYTES);


	// Execute the kernel
	what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_warp, gpu_cal_thread);

	// copy back the gpu results to the cpu
	hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_calc_thread, gpu_cal_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);


	// free the arrays on the GPU 
	hipFree(gpu_block);
	hipFree(gpu_thread);
	hipFree(gpu_warp);
	hipFree(gpu_cal_thread);

	// print 
	for (i=0; i<ARRAY_SIZE; i++)
	{
		printf("Calculated Thread: %d - Block: %d - Warp %d - Thread %d \n",
				cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);
	}

}
