#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

typedef float FLOAT;
#define USE_UNIX 1

/* get thread id: 1D block and 2D grid */
#define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x)

/* get block id: 2D grid */
#define get_bid() (blockIdx.x + blockIdx.y * gridDim.x)

/* warm up, start GPU, optional */
void warmup();

/* get time stamp */
double get_time(void);

/* host, add */
void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N);

/* device function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 1D block */
    int idx = get_tid();

    if (idx < N) z[idx] = z[idx] + y[idx] + x[idx];
}

void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    int i;

    for (i = 0; i < N; i++) z[i] = z[i] + y[i] + x[i];
}

/* a little system programming */
#if USE_UNIX
#include <sys/time.h>
#include <time.h>

double get_time(void)
{
    struct timeval tv;
    double t;

    gettimeofday(&tv, (struct timezone *)0);
    t = tv.tv_sec + (double)tv.tv_usec * 1e-6;

    return t;
}
#else
#include <windows.h>

double get_time(void)
{
    LARGE_INTEGER timer;
    static LARGE_INTEGER fre;
    static int init = 0;
    double t;

    if (init != 1) {
        QueryPerformanceFrequency(&fre);
        init = 1;
    }

    QueryPerformanceCounter(&timer);

    t = timer.QuadPart * 1. / fre.QuadPart;

    return t;
}
#endif

/* warm up GPU */
__global__ void warmup_knl()
{
    int i, j;

    i = 1;
    j = 2;
    i = i + j;
}

void warmup()
{
    int i;

    for (i = 0; i < 8; i++) {
        warmup_knl<<<1, 256>>>();
    }
}

int main()
{
    int N = 20000000;  // we have N threads
    int nbytes = N * sizeof(FLOAT);  // total bytes

    /* 1D block */
    int bs = 256; // each block has 256 threads, so we need N / 256 blocks

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));  // we use 2D grid, with dim sqrt(N/256)
    dim3 grid = dim3(s, s);

    /* init the variable */
    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL, *hy = NULL;
    FLOAT *dz = NULL, *hz = NULL;

    int itr = 30;
    int i;
    double th, td; 

    /* warm up GPU */
    warmup();

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    if (dx == NULL || dy == NULL || dz == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %.2f MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);
    hz = (FLOAT *) malloc(nbytes);

    if (hx == NULL || hy == NULL || hz == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %.2f MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
        hz[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dz, hz, nbytes, hipMemcpyHostToDevice);

    /* warm up */
    warmup();

    /* call GPU */
    hipDeviceSynchronize();
    td = get_time();
    
    for (i = 0; i < itr; i++) vec_add<<<grid, bs>>>(dx, dy, dz, N);

    hipDeviceSynchronize();
    td = get_time() - td;

    /* CPU */
    th = get_time();
    for (i = 0; i < itr; i++) vec_add_host(hx, hy, hz, N);
    th = get_time() - th;

    printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);

    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);

    return 0;
}
