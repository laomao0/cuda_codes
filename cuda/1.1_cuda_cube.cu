
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void cube(float * d_out, float * d_in){
    int tid = threadIdx.x;

    d_out[tid] = d_in[tid] * d_in[tid] * d_in[tid];

}

int main(int argc, char ** arhv){
    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    //generate the input array on the host
    float h_in[ARRAY_SIZE];
    for( int i = 0; i < ARRAY_SIZE; i++){
        h_in[i] = float(i);
    }

    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float * d_in;
    float * d_out;

    // Allocate GPU Memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    //print out result
    for(int i = 0; i < ARRAY_SIZE; i++){
        printf("%f", h_out[i]);
        printf(((i % 4) != 3) ? "\t": "\n");
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;

}