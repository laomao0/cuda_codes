#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.
#define BLOCKDIMX (32)
#define BLOCKDIMY (16)
#define CHANNEL (1)
#include "utils.h"
#include <iostream>


__global__
void rbga_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset

  const int DIM_GRID_X = gridDim.x;
  const int DIM_GRID_Y = gridDim.y;
  const int DIM_BLOCK_X = blockDim.x;
  const int DIM_BLOCK_Y = blockDim.y;

  const int w_i = blockIdx.x * DIM_BLOCK_X + threadIdx.x;
  const int h_i = blockIdx.y * DIM_BLOCK_Y + threadIdx.y;

  const bool w_bound = w_i < numCols;
  const bool h_bound = h_i < numRows;

  int offset = h_i * numCols + w_i;

  if ( w_bound && h_bound){

    uchar4 data4 = rgbaImage[offset];

    float R = data4.x;
    float G = data4.y;
    float B = data4.z;

    float greyness = .299f * R + .587f * G + .114f * B;

    greyImage[offset] = static_cast<unsigned char>(greyness);  // float to uchar

  }
  
  
}



void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(BLOCKDIMX, BLOCKDIMY, 1);  //TODO
  const dim3 gridSize( (numCols + BLOCKDIMX -1) / BLOCKDIMX, (numRows + BLOCKDIMY -1) / BLOCKDIMY, CHANNEL );  //TODO

  std::cout << "blockSize " << BLOCKDIMX << ","<< BLOCKDIMY << ",1" << std::endl;
  std::cout << "gridSize " << (numCols + BLOCKDIMX -1) / BLOCKDIMX << ","
                           << (numRows + BLOCKDIMY -1) / BLOCKDIMY << ","
                           << CHANNEL << std::endl;
 
  rbga_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());

}
