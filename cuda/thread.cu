#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

//thread 1D

__global__ void testThread1(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = b[i] - a[i];
}

//thread 2D
__global__ void testThread2(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + threadIdx.y*blockDim.x;

    // printf("Block Idx X %d \n", blockIdx.x); // 0, only 1 grid, 1 block

    c[i] = b[i] - a[i];
    if(i == 0){
        printf("Block Dim X %d \n", blockDim.x); // 200
        printf("Block Dim Y %d \n", blockDim.y); // 5
    }

}

//thread 3D
__global__ void testThread3(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    c[i] = b[i] - a[i];
}

//block 1D
__global__ void testBlock1(int *c, const int *a, const int *b)
{
    int i = blockIdx.x;
    c[i] = b[i] - a[i];
}

//block 2D
__global__ void testBlock2(int *c, const int *a, const int *b)
{
    int i = blockIdx.x + blockIdx.y*gridDim.x;
    c[i] = b[i] - a[i];
}

//block 3D
__global__ void testBlock3(int *c, const int *a, const int *b)
{
    int i = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    c[i] = b[i] - a[i];
}

//block-thread 1D-1D
__global__ void testBlockThread1(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    c[i] = b[i] - a[i];
}

//block-thread 1D-2D
__global__ void testBlockThread2(int *c, const int *a, const int *b)
{
    int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    int i = threadId_2D+ (blockDim.x*blockDim.y)*blockIdx.x;
    c[i] = b[i] - a[i];
}

//block-thread 1D-3D
__global__ void testBlockThread3(int *c, const int *a, const int *b)
{
    int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockIdx.x;
    c[i] = b[i] - a[i];
}

//block-thread 2D-1D
__global__ void testBlockThread4(int *c, const int *a, const int *b)
{
    int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    int i = threadIdx.x + blockDim.x*blockId_2D;
    c[i] = b[i] - a[i];
}

//block-thread 3D-1D
__global__ void testBlockThread5(int *c, const int *a, const int *b)
{
    int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int i = threadIdx.x + blockDim.x*blockId_3D;
    c[i] = b[i] - a[i];
}

//block-thread 2D-2D
__global__ void testBlockThread6(int *c, const int *a, const int *b)
{
    int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    int i = threadId_2D + (blockDim.x*blockDim.y)*blockId_2D;
    c[i] = b[i] - a[i];
}

//block-thread 2D-3D
__global__ void testBlockThread7(int *c, const int *a, const int *b)
{
    int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockId_2D;
    c[i] = b[i] - a[i];
}

//block-thread 3D-2D
__global__ void testBlockThread8(int *c, const int *a, const int *b)
{
    int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int i = threadId_2D + (blockDim.x*blockDim.y)*blockId_3D;
    c[i] = b[i] - a[i];
}

//block-thread 3D-3D
__global__ void testBlockThread9(int *c, const int *a, const int *b)
{
    int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockId_3D;
    c[i] = b[i] - a[i];
}


void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // testThread1<<<1, size>>>(dev_c, dev_a, dev_b);

    // uint3 s;s.x = size/5;s.y = 5;s.z = 1;
    // testThread2 <<<1,s>>>(dev_c, dev_a, dev_b);

    // uint3 s; s.x = size / 10; s.y = 5; s.z = 2;
    // testThread3<<<1, s >>>(dev_c, dev_a, dev_b);

    // testBlock1<<<size,1 >>>(dev_c, dev_a, dev_b);

    //uint3 s; s.x = size / 5; s.y = 5; s.z = 1;
    // testBlock2<<<s, 1 >>>(dev_c, dev_a, dev_b);

    //uint3 s; s.x = size / 10; s.y = 5; s.z = 2;
    //testBlock3<<<s, 1 >>>(dev_c, dev_a, dev_b);

    // usually used in cupy 
    // https://github.com/laomao0/cupy_packages/blob/main/Interpolation/interpolation_cupy.py
    //testBlockThread1<<<size/10, 10>>>(dev_c, dev_a, dev_b);

    // uint3 s1; s1.x = size / 100; s1.y = 1; s1.z = 1;
    // uint3 s2; s2.x = 10; s2.y = 10; s2.z = 1;
    // testBlockThread2 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    // uint3 s1; s1.x = size / 100; s1.y = 1; s1.z = 1;
    // uint3 s2; s2.x = 10; s2.y = 5; s2.z = 2;
    // testBlockThread3 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1; s1.x = 10; s1.y = 10; s1.z = 1;
    //uint3 s2; s2.x = size / 100; s2.y = 1; s2.z = 1;
    //testBlockThread4 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1; s1.x = 10; s1.y = 5; s1.z = 2;
    //uint3 s2; s2.x = size / 100; s2.y = 1; s2.z = 1;
    //testBlockThread5 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1; s1.x = size / 100; s1.y = 10; s1.z = 1;
    //uint3 s2; s2.x = 5; s2.y = 2; s2.z = 1;
    //testBlockThread6 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1; s1.x = size / 100; s1.y = 5; s1.z = 1;
    //uint3 s2; s2.x = 5; s2.y = 2; s2.z = 2;
    //testBlockThread7 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    // usually used in tradational cuda codes
    // https://github.com/laomao0/cupy_packages/blob/main/Interpolation/interpolation_cuda_kernel.cu
    // for a image with batch-chw size, block size typically 32,16,1
    // block  = dim3(BLOCKDIMX,BLOCKDIMY,1);
    // grid = dim3( (w + BLOCKDIMX - 1)/ BLOCKDIMX, (h + BLOCKDIMY - 1) / BLOCKDIMY, batch);
    uint3 s1; s1.x = 5; s1.y = 2; s1.z = 2;
    uint3 s2; s2.x = size / 100; s2.y = 5; s2.z = 1;
    testBlockThread8 <<<s1, s2 >>>(dev_c, dev_a, dev_b);

    // uint3 s1; s1.x = 5; s1.y = 2; s1.z = 2;
    // uint3 s2; s2.x = size / 200; s2.y = 5; s2.z = 2;
    // testBlockThread9<<<s1, s2 >>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipGetLastError();
}


int main()
{
    const int n = 1000;

    int *a = new int[n];
    int *b = new int[n];
    int *c = new int[n];
    int *cc = new int[n];

    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
        c[i] = b[i] - a[i];  // cpu value
    }

    addWithCuda(cc, a, b, n);

    FILE *fp = fopen("out.out", "w");
    for (int i = 0; i < n; i++)
        fprintf(fp, "%d %d\n", c[i], cc[i]);
    fclose(fp);

    bool flag = true;
    for (int i = 0; i < n; i++)
    {
        if (c[i] != cc[i])
        {
            flag = false;
            break;
        }
    }

    if (flag == false)
        printf("no pass");
    else
        printf("pass");

    hipDeviceReset();

    delete[] a;
    delete[] b;
    delete[] c;
    delete[] cc;

    getchar();
    return 0;
}
